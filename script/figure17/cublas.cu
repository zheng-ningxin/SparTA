////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

//
// Matrix multiplication: C = A * B.
// Host code.
//
// This sample implements matrix multiplication as described in Chapter 3
// of the programming guide and uses the CUBLAS library to demonstrate
// the best performance.

// SOME PRECAUTIONS:
// IF WE WANT TO CALCULATE ROW-MAJOR MATRIX MULTIPLY C = A * B,
// WE JUST NEED CALL CUBLAS API IN A REVERSE ORDER: cublasSegemm(B, A)!
// The reason is explained as follows:

// CUBLAS library uses column-major storage, but C/C++ use row-major storage.
// When passing the matrix pointer to CUBLAS, the memory layout alters from
// row-major to column-major, which is equivalent to an implicit transpose.

// In the case of row-major C/C++ matrix A, B, and a simple matrix multiplication
// C = A * B, we can't use the input order like cublasSgemm(A, B)  because of
// implicit transpose. The actual result of cublasSegemm(A, B) is A(T) * B(T).
// If col(A(T)) != row(B(T)), equal to row(A) != col(B), A(T) and B(T) are not
// multipliable. Moreover, even if A(T) and B(T) are multipliable, the result C
// is a column-based cublas matrix, which means C(T) in C/C++, we need extra
// transpose code to convert it to a row-based C/C++ matrix.

// To solve the problem, let's consider our desired result C, a row-major matrix.
// In cublas format, it is C(T) actually (because of the implicit transpose).
// C = A * B, so C(T) = (A * B) (T) = B(T) * A(T). Cublas matrice B(T) and A(T)
// happen to be C/C++ matrice B and A (still because of the implicit transpose)!
// We don't need extra transpose code, we only need alter the input order!
//
// CUBLAS provides high-performance matrix multiplication.
// See also:
// V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
// in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
// Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
//

// Utilities and system includes
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on CPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
void
matrixMulCPU(float *C, const float *A, const float *B, unsigned int M, unsigned int K, unsigned int N)
{
    for(int i = 0; i < M; i += 1){
        for(int j = 0; j < N; j += 1){
            double sum = 0;
            for(int k = 0; k < K; k += 1){
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = (float)sum;
        }
    }
}

void init(float * ptr, size_t length, float sparsity)
{
    // lock the random seed for
    srand (1);
    for (int i = 0; i < length; i++)
    {
        float pro = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        //printf("pro: %f\n", pro);
        if (pro < sparsity)
        {
            ptr[i] = 0.0;
        }
        else
        {
            ptr[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test matrix multiply using CUBLAS
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char*argv[]) {
    float sparsity_ratio = atof(argv[1]);
    printf("Sparsity Ratio=%f\n", sparsity_ratio);

    const int M = 1024;
    const int K = 1024;
    const int N = 1024;

    int block_size = 32;

    // allocate host memory for matrices A and B
    unsigned int size_A = M * K;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = K * N;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    init(h_A, size_A, sparsity_ratio);
    init(h_B, size_B, 0);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    unsigned int size_C = M * N;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    float *h_C      = (float *) malloc(mem_size_C);
    float *h_CUBLAS = (float *) malloc(mem_size_C);

    checkCudaErrors(hipMalloc((void **) &d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void **) &d_B, mem_size_B));
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &d_C, mem_size_C));

    // create and start timer
    printf("Computing result using CUBLAS...");

    // execute the kernel
    int nIter = 30;

    // CUBLAS version 2.0
    {
        const float alpha = 1.0f;
        const float beta  = 0.0f;
        hipblasHandle_t handle;
        hipEvent_t start, stop;

        hipblasCreate(&handle);

        //Perform warmup operation with cublas
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N);

        // Allocate CUDA events that we'll use for timing
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));

        // Record the start event
        checkCudaErrors(hipEventRecord(start, NULL));

        for (int j = 0; j < nIter; j++)
        {
            //note cublas is column primary!
            //need to transpose the order
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N);

        }

        // Record the stop event
        checkCudaErrors(hipEventRecord(stop, NULL));

        // Wait for the stop event to complete
        checkCudaErrors(hipEventSynchronize(stop));

        printf("done.\n");

        float msecTotal = 0.0f;
        checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

        // Compute and print the performance
        float msecPerMatrixMul = msecTotal / nIter;
        double flopsPerMatrixMul = 2.0 * (double)M * (double)N * (double)K;
        double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
        printf(
            "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
            gigaFlops,
            msecPerMatrixMul,
            flopsPerMatrixMul);

        // copy result from device to host
        checkCudaErrors(hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost));

        // Destroy the handle
        hipblasDestroy(handle);
    }

    // compute reference solution
    printf("Computing result using host CPU...");
    float *reference = (float *)malloc(mem_size_C);
    matrixMulCPU(reference, h_A, h_B, M, K, N);
    printf("done.\n");

    bool correct = true;
    double eps = 1.e-6;
    for(int i = 0; i < M * N; i++){
        double abs_err = abs(reference[i] - h_CUBLAS[i]);
        double dot_length = M;
        double abs_val = abs(h_CUBLAS[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%lf, ref=%lf error term is > %E\n",
                    i, h_CUBLAS[i], reference[i], eps);
            correct = false;
            break;
        }
    }

    if(correct) printf("Result = Pass\n");
    else printf("Result = Fail\n");

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));


    return EXIT_SUCCESS;    // return value = 1
}
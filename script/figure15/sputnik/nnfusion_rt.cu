#include <sstream>
#include <stdexcept>
#include <hipDNN.h>
#include <hipblas.h>
#include "sputnik/cuda_utils.h"
#include "sputnik/matrix_utils.h"
#include "sputnik/spmm/cuda_spmm.h"
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <fstream>
#include "nnfusion_rt.h"
#include <limits>
#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}
extern "C" int kernel_entry(int m, int k, int n, int nnz, int*d_row_swizzle, float*d_values, int*d_row_idx, int*d_col_idx, float* dB, float * dC, int beta){
    CHECK_CUDA(sputnik::CudaSpmm(m, k, n, nnz, d_row_swizzle, d_values, d_row_idx, d_col_idx, dB, dC, 0);)
}
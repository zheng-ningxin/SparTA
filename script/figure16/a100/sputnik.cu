#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <cusparseLt.h>       // cusparseLt header
#include <cstdio>             // printf
#include <cstdlib>            // std::rand
#include "sputnik/cuda_utils.h"
#include "sputnik/matrix_utils.h"
#include "sputnik/spmm/cuda_spmm.h"
#include "time.h"
#include <vector>
using namespace std;
using namespace sputnik;
#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

constexpr int EXIT_UNSUPPORTED = 2;
// for finegrained kernels
int32_t * row_idx, *col_idx, *d_row_idx, *d_col_idx, *row_swizzle, *d_row_swizzle;
int32_t row_idx_size, col_idx_size, values_size;
float * values, *d_values;
void init(float * ptr, size_t length, float sparsity)
{
    // lock the random seed for
    srand (1);
    for (int i = 0; i < length; i++)
    {
        float pro = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        //printf("pro: %f\n", pro);
        if (pro < sparsity)
        {
            ptr[i] = 0.0;
        }
        else
        {
            ptr[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        }
    }
}
void SortedRowSwizzle(int rows, int *row_offsets, int *row_indices) {
  // Create our unsorted row indices.
  std::vector<int> swizzle_staging(rows);
  std::iota(swizzle_staging.begin(), swizzle_staging.end(), 0);

  // Argsort the row indices based on their length.
  std::sort(swizzle_staging.begin(), swizzle_staging.end(),
            [&row_offsets](int idx_a, int idx_b) {
              int length_a = row_offsets[idx_a + 1] - row_offsets[idx_a];
              int length_b = row_offsets[idx_b + 1] - row_offsets[idx_b];
              return length_a > length_b;
            });

  // Copy the ordered row indices to the output.
  std::memcpy(row_indices, swizzle_staging.data(), sizeof(int) * rows);
}

int convert_csr(float * ptr, int32_t row, int32_t col, int32_t * &row_idx, int32_t * &col_idx, float * &values)
{
    auto v_row_idx = std::make_shared<vector<int32_t>>();
    auto v_col_idx = std::make_shared<vector<int32_t>>();
    auto v_values = std::make_shared<vector<float>>();

    for (int i = 0; i < row; i++)
    {
        v_row_idx->push_back(v_values->size());
        for (int j = 0; j < col; j++)
        {
            size_t pos = i * col + j;
            if (ptr[pos] < 1e-8)
            {
                // sparsity
                continue;
            }
            else
            {
                v_values->push_back(ptr[pos]);
                v_col_idx->push_back(j);
            }
        }
    }
    v_row_idx->push_back(v_values->size());
    row_idx_size = sizeof(int32_t)*v_row_idx->size();
    col_idx_size = sizeof(int32_t)*v_col_idx->size();
    values_size = sizeof(float)*v_values->size();
    printf("values_size: %d\n", values_size);
    row_idx = (int32_t*) malloc(row_idx_size);
    col_idx = (int32_t*) malloc(col_idx_size);
    values = (float*) malloc(values_size);
    memcpy(row_idx, v_row_idx->data(), row_idx_size);
    memcpy(col_idx, v_col_idx->data(), col_idx_size);
    memcpy(values, v_values->data(), values_size);
    return v_values->size();
}

void transform(float* A, float*A1, float*A2, int length){
    // split the matrix A into A1 and A2
    // A1 is for the saprse tensor core, A2 is for the finegrained sparse kernel
    memset(A1, 0, sizeof(float)*length);
    memset(A2, 0, sizeof(float)*length);
    assert(length%2==0);
    int nnz=0;
    for(int i=0; i<length/2;i++){
        int start = i*2;
        int end= start+2;
        nnz=0;
        for(int j=start; j<end; j++){
            if(A[j]!=0){
                if(nnz<1){
                    A1[j]=A[j];
                }else{
                    A2[j]=A[j];
                }
                nnz++;
            }
        }
    }
}

int main(int argc, char*argv[]) {
    float sparsity_ratio = atof(argv[1]);
    printf("Sparsity Ratio=%f\n", sparsity_ratio);
    int major_cc, minor_cc;
    // Host problem definition, row-major order
    constexpr int m     = 1024; // bigger sizes may require dynamic allocations
    constexpr int n     = 1024; // bigger sizes may require dynamic allocations
    constexpr int k     = 1024; // bigger sizes may require dynamic allocations
    auto          order = HIPSPARSE_ORDER_ROW;
    auto          opA   = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          opB   = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          type  = HIP_R_32F;
    auto          compute_type = CUSPARSE_COMPUTE_TF32;

    bool     is_rowmajor    = (order == HIPSPARSE_ORDER_ROW);
    bool     isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    bool     isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    auto     num_A_rows     = (isA_transposed) ? k : m;
    auto     num_A_cols     = (isA_transposed) ? m : k;
    auto     num_B_rows     = (isB_transposed) ? n : k;
    auto     num_B_cols     = (isB_transposed) ? k : n;
    auto     num_C_rows     = m;
    auto     num_C_cols     = n;
    unsigned alignment      = 16;
    auto     lda            = (is_rowmajor) ? num_A_cols : num_A_rows;
    auto     ldb            = (is_rowmajor) ? num_B_cols : num_B_rows;
    auto     ldc            = (is_rowmajor) ? num_C_cols : num_C_rows;
    auto     A_height       = (is_rowmajor) ? num_A_rows : num_A_cols;
    auto     B_height       = (is_rowmajor) ? num_B_rows : num_B_cols;
    auto     C_height       = (is_rowmajor) ? num_C_rows : num_C_cols;
    auto     A_size         = A_height * lda * sizeof(float);
    auto     B_size         = B_height * ldb * sizeof(float);
    auto     C_size         = C_height * ldc * sizeof(float);
    float hA[m * k];
    float hB[k * n];
    float hC[m * n] = {};

    init(hA, m*k, sparsity_ratio);
    init(hB, k*n, 0);
    
    // build the index for the finegrained kernel
    convert_csr(hA, m,k, row_idx, col_idx, values);
    CHECK_CUDA(hipMalloc(&d_row_idx, row_idx_size));
    CHECK_CUDA(hipMalloc(&d_col_idx, col_idx_size));
    CHECK_CUDA(hipMalloc(&d_values, values_size));
    CHECK_CUDA(hipMemcpy(d_row_idx, row_idx, row_idx_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_col_idx, col_idx, col_idx_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_values, values, values_size, hipMemcpyHostToDevice));
    row_swizzle = (int *) malloc(sizeof(int) * m);
    CHECK_CUDA(hipMalloc(&d_row_swizzle, sizeof(int)*m));
    SortedRowSwizzle(m, row_idx, row_swizzle);
    CHECK_CUDA(hipMemcpy(d_row_swizzle, row_swizzle, sizeof(int)*m, hipMemcpyHostToDevice));
    int fine_nnz = values_size / sizeof(float);
    printf("fine_nnz: %d\n", fine_nnz);
    float alpha = 1.0f;
    float beta  = 1.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    float *dA, *dB, *dC, *dD;
    CHECK_CUDA( hipMalloc((void**) &dA, A_size) )
    CHECK_CUDA( hipMalloc((void**) &dB, B_size) )
    CHECK_CUDA( hipMalloc((void**) &dC, C_size) )
    
    CHECK_CUDA( hipMemcpy(dA, hA, A_size, hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size, hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC, hC, C_size, hipMemcpyHostToDevice) )
    
    //--------------------------------------------------------------------------
    
    float ms_total;
    int n_iter = 1000;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for(int i=0;i<n_iter;i++){
        CHECK_CUDA(CudaSpmm(m ,k, n, fine_nnz, d_row_swizzle, d_values, d_row_idx, d_col_idx, dB, dC, 0));

    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms_total, start, stop);
    printf("Timecost: %f ms\n",ms_total/n_iter);
    //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
    //CHECK_CUDA( hipMemcpy(hA, dA, A_size, hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hC, dC, C_size, hipMemcpyDeviceToHost) )

    bool A_std_layout = (is_rowmajor != isA_transposed);
    bool B_std_layout = (is_rowmajor != isB_transposed);
    // host computation
    float hC_result[m * n];
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            float sum  = 0.0f;
            for (int k1 = 0; k1 < k; k1++) {
                auto posA = (A_std_layout) ? i * lda + k1 : i + k1 * lda;
                auto posB = (B_std_layout) ? k1 * ldb + j : k1 + j * ldb;
                sum      += static_cast<float>(hA[posA]) *  // [i][k]
                            static_cast<float>(hB[posB]);   // [k][j]
            }
            auto posC       = (is_rowmajor) ? i * ldc + j : i + j * ldc;
            //printf("sum:%f \n",sum);
        hC_result[posC] = sum;  // [i][j]
        }
    }
    // host-device comparison
    int correct = 1;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            auto pos          = (is_rowmajor) ? i * ldc + j : i + j * ldc;
            auto device_value = static_cast<float>(hC[pos]);
            auto host_value   = hC_result[pos];
            if (fabs(device_value - host_value)/host_value>1e-3) {
                // direct floating point comparison is not reliable
                std::printf("(%d, %d):\t%f vs. %f\n",
                            i, j, host_value, device_value);
                correct = 0;
                break;
            }
        }
    }
    if (correct)
        std::printf("spmma_example test PASSED\n");
    else
        std::printf("spmma_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC) )
    return EXIT_SUCCESS;
}

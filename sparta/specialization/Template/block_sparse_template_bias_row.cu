#include "hip/hip_runtime.h"



__global__ void BLOCK_SPARSE_MATMUL(float* input0, float* input1,float* input2, float* input3, float* input4, float *output0){

    const int BLOCK_SIZE_M=BLOCK_SIZE_M_VALUE;
    const int BLOCK_SIZE_K=BLOCK_SIZE_K_VALUE;
    const int BLOCK_SIZE_N=BLOCK_SIZE_N_VALUE;
    const int THREAD_SIZE_M=THREAD_SIZE_M_VALUE;
    const int THREAD_SIZE_K=THREAD_SIZE_K_VALUE;
    const int THREAD_SIZE_N=THREAD_SIZE_N_VALUE;
    const int M=M_VALUE;
    const int N=N_VALUE;
    const int K=K_VALUE;
    float * A = reinterpret_cast<float*>(input0);
    float * W_val = reinterpret_cast<float*>(input1);
    int * W_row = reinterpret_cast<int*>(input2);
    int * W_col = reinterpret_cast<int*>(input3);
    float * bias = reinterpret_cast<float*>(input4);
    float * C = reinterpret_cast<float*>(output0);
    /* 
    COMMENT_TAG
    */
    int by = blockIdx.y;
    int bx = blockIdx.x;
    int ty = threadIdx.y;
    int tx = threadIdx.x;

    __shared__ float As[BLOCK_SIZE_M * BLOCK_SIZE_K];
    __shared__ float Bs[BLOCK_SIZE_N * BLOCK_SIZE_K];

    float accum[THREAD_SIZE_N][THREAD_SIZE_M] = {0};
    float a_frag[THREAD_SIZE_M][THREAD_SIZE_K];
    float b_frag[THREAD_SIZE_N][THREAD_SIZE_K];

    int A_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    int B_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    int bszy = BLOCK_SIZE_M / THREAD_SIZE_M;
    int bszx = BLOCK_SIZE_N / THREAD_SIZE_N;

    int THREADS_PER_BLOCK = bszy * bszx;

    int A_TILE_ROW_STRIDE = THREADS_PER_BLOCK / A_THREAD_PER_ROW;
    int B_TILE_ROW_STRIDE = THREADS_PER_BLOCK / B_THREAD_PER_ROW;

    int tid = ty * bszx + tx;

    int A_BLOCK_ROW_START = tid / A_THREAD_PER_ROW;
    int B_BLOCK_ROW_START = tid / B_THREAD_PER_ROW;

    int A_BLOCK_COL_START = tid % A_THREAD_PER_ROW * 4;
    int B_BLOCK_COL_START = tid % B_THREAD_PER_ROW * 4;

    int index_start = W_row[bx], index_end = W_row[bx+1];

    const int vBLOCK_SIZE_M = BLOCK_SIZE_M / THREAD_SIZE_M;
    const int vBLOCK_SIZE_N = BLOCK_SIZE_N / THREAD_SIZE_N;
    for(int tile_block_idx = index_start; tile_block_idx < index_end; tile_block_idx += 1){
        int tile_idx = W_col[tile_block_idx] * BLOCK_SIZE_K;
        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_M; k += A_TILE_ROW_STRIDE){
            FETCH_FLOAT4(As[OFFSET(k+A_BLOCK_ROW_START, A_BLOCK_COL_START, BLOCK_SIZE_K)]) =
                FETCH_FLOAT4(A[OFFSET(by*BLOCK_SIZE_M+k+A_BLOCK_ROW_START, tile_idx+A_BLOCK_COL_START, K)]);
        }
        /*
        for(int k = 0; k < BLOCK_SIZE_K; k += A_TILE_ROW_STRIDE){
            FETCH_FLOAT4(As[OFFSET(k+A_BLOCK_ROW_START, A_BLOCK_COL_START, BLOCK_SIZE_M)]) = 
                FETCH_FLOAT4(A[OFFSET(tile_idx+k+A_BLOCK_ROW_START, by*BLOCK_SIZE_M+A_BLOCK_COL_START, M)]);
        }
        */

        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_K; k += B_TILE_ROW_STRIDE){
            FETCH_FLOAT4(Bs[OFFSET(k+B_BLOCK_ROW_START, B_BLOCK_COL_START, BLOCK_SIZE_N)]) = 
                FETCH_FLOAT4(W_val[tile_block_idx * BLOCK_SIZE_N * BLOCK_SIZE_K + (k+B_BLOCK_ROW_START) * BLOCK_SIZE_N + B_BLOCK_COL_START]);
                // FETCH_FLOAT4(B[OFFSET(tile_idx+k+B_BLOCK_ROW_START, bx*BLOCK_SIZE_N+B_BLOCK_COL_START, N)]);
        }

        __syncthreads();

        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_K; k += THREAD_SIZE_K){
            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_K; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_M; j += 1){
                    a_frag[j][i] = As[OFFSET(ty + vBLOCK_SIZE_M * j, k+i, BLOCK_SIZE_K)];
                    //a_frag[j][i] = As[OFFSET(k+i, ty + vBLOCK_SIZE_M * j, BLOCK_SIZE_M)];
                }
            }

            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_K; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_N; j += 1){
                    b_frag[j][i] = Bs[OFFSET(k+i, tx + vBLOCK_SIZE_N * j, BLOCK_SIZE_N)];
                }
            }

            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_N; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_M; j++){
                    #pragma unroll
                    for(int k_in = 0; k_in < THREAD_SIZE_K; k_in++){
                        // accum[i][j] = fma(a_frag[j][k_in], b_frag[i][k_in], accum[i][j]);
                        accum[i][j] += a_frag[j][k_in] * b_frag[i][k_in];
                    }
                }
            }
        }

        __syncthreads();
    }

    float bias_local[THREAD_SIZE_N];
    for(int thread_x = 0; thread_x < THREAD_SIZE_N; thread_x++){
        bias_local[thread_x] = bias[BLOCK_SIZE_N * bx + tx + thread_x * vBLOCK_SIZE_N];
    }

    #pragma unroll
    for(int thread_x = 0; thread_x < THREAD_SIZE_N; thread_x++){
        #pragma unroll
        for(int thread_y = 0; thread_y < THREAD_SIZE_M; thread_y+=1){
            C[OFFSET(
                BLOCK_SIZE_M * by + ty + thread_y * vBLOCK_SIZE_M,
                BLOCK_SIZE_N * bx + tx + thread_x * vBLOCK_SIZE_N,
                N
            )] = (accum[thread_x][thread_y]) + bias_local[thread_x];
        }
    }
}
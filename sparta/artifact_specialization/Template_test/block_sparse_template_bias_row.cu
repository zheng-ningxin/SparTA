#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <cstring>
// CUDA runtime
#include <hip/hip_runtime.h>

#ifndef BCSR_H
#define BCSR_H

#include <fstream>
#include <iostream>

using namespace std;

class bcsr {
public:
    float* val;
    int* is_block_present;
    int* col_idx;
    int* row_ptr;
    int m, n, m_block_sz, n_block_sz, m_block, n_block, nnz_block_num;

    bcsr(int m, int n, int m_block_sz, int n_block_sz): m(m), n(n), m_block_sz(m_block_sz), n_block_sz(n_block_sz) {
        m_block = m / m_block_sz;       // number of K axis block
        n_block = n / n_block_sz;       // number of N axis block
        nnz_block_num = 0;
        is_block_present = (int*) malloc(sizeof(int) * m_block * n_block);
        val = NULL;
        col_idx = NULL;
        row_ptr = NULL;
    }
    
    ~bcsr() {
        free(is_block_present);
        if (val != NULL) free(val);
        if (col_idx != NULL) free(col_idx);
        if (row_ptr != NULL) free(row_ptr);
    }

    void print() {
        printf("is block present: \n");
        for ( int i = 0 ; i < m_block ; i ++ ) {
            for ( int j = 0 ; j < n_block ; j ++ ) {
                printf("%d ", is_block_present[i * n_block + j]);
            }
            printf("\n");
        }

        printf("row_ptr: \n");
        for ( int i = 0 ; i < m_block + 1 ; i ++ ) {
            printf("%d ", row_ptr[i]);
        }
        printf("\n");
        printf("col_idx: \n");
        for ( int i = 0 ; i < nnz_block_num ; i ++ ) {
            printf("%d ", col_idx[i]);
        }
    }

    void load_val(){
        size_t size_val = nnz_block_num * m_block_sz * n_block_sz;
        size_t size_row = m_block + 1;
        size_t size_col = nnz_block_num;
        ifstream f_val("val.bin", ios::out | ios::binary);
        ifstream f_row("row.bin", ios::out | ios::binary);
        ifstream f_col("col.bin", ios::out | ios::binary);
        f_val.read((char *)val, sizeof(float) * size_val);
        f_row.read((char *)row_ptr, sizeof(int) * size_row);
        f_col.read((char *)col_idx, sizeof(int) * size_col);
        f_val.close();
        f_row.close();
        f_col.close();
    }

    void export_val(){
        size_t size_val = nnz_block_num * m_block_sz * n_block_sz;
        size_t size_row = m_block + 1;
        size_t size_col = nnz_block_num;
        ofstream f_val("val.bin", ios::out | ios::binary);
        ofstream f_row("row.bin", ios::out | ios::binary);
        ofstream f_col("col.bin", ios::out | ios::binary);
        f_val.write((char *)val, sizeof(float) * size_val);
        f_row.write((char *)row_ptr, sizeof(int) * size_row);
        f_col.write((char *)col_idx, sizeof(int) * size_col);
        f_val.close();
        f_row.close();
        f_col.close();
    }
};
#endif

#ifndef UTILS_H
#define UTILS_H

#include <stdio.h>
#include <stdint.h>

void cal_block(bcsr*, float* );
void generate_bcsr(bcsr*, float* );

void cal_block(bcsr* mat, float* data) {
    // m_block : number of K axis block
    // n_block : number of N axis block
    for ( int i = 0 ; i < mat->m_block * mat->n_block ; i ++ ) {
        mat->is_block_present[i] = 0;
    }
    for ( int i = 0 ; i < mat->m * mat->n ; i ++ ) {
        if (data[i] != 0) {
            // 计算属于哪一个block
            int m_block_idx = i / mat->n / mat->m_block_sz;     // block index of K axis
            int n_block_idx = i % mat->n / mat->n_block_sz;     // block index of N axis
            /*
            if (mat->is_block_present[m_block_idx * mat->n_block + n_block_idx] == 0) {
                mat->is_block_present[m_block_idx * mat->n_block + n_block_idx] = 1;
                mat->nnz_block_num += 1;
            }
            */
            if (mat->is_block_present[n_block_idx * mat->m_block + m_block_idx] == 0) {
                mat->is_block_present[n_block_idx * mat->m_block + m_block_idx] = 1;
                mat->nnz_block_num += 1;
            }
        }
    }
}

void generate_bcsr(bcsr* mat, float* data) {
    int ptr = 0;
    int block_ptr = 0;
    int row_ptr = 0;
    mat->row_ptr[row_ptr ++ ] = block_ptr;
    for( int i = 0; i < mat->n_block; i += 1){
        for( int j = 0; j < mat->m_block; j += 1){
            if(mat->is_block_present[i * mat->m_block + j] == 1){
                mat->col_idx[block_ptr ++] = j;
                for (int i_block = 0; i_block < mat->m_block_sz; i_block += 1){
                    for(int j_block = 0; j_block < mat->n_block_sz; j_block += 1){
                        mat->val[ptr++] = data[(j * mat->m_block_sz+i_block) * mat->n + (i * mat->n_block_sz + j_block)];
                    }
                }
            }
        }
        mat->row_ptr[row_ptr ++] = block_ptr;
    }
    /*
    for ( int i = 0 ; i < mat->m_block ; i += 1) {
        for ( int j = 0 ; j < mat->n_block ; j += 1) {
            if ( mat->is_block_present[i * mat->n_block + j] == 1) {
                mat->col_idx[block_ptr ++ ] = j;
                // copy whole block into val
                for (int i_block = 0 ; i_block < mat->m_block_sz ; i_block ++ ) {
                    for ( int j_block = 0 ; j_block < mat->n_block_sz ; j_block ++) {
                        mat->val[ptr ++ ] = data[ (i * mat->m_block_sz + i_block) * mat->n + (j * mat->n_block_sz + j_block)];
                    }
                }
            }
        }
        // 记录row_ptr
        mat->row_ptr[row_ptr ++ ] = block_ptr;
    }
    */
}

#endif

#define OFFSET(row, col, ld) ((row) * ld + col)

#define CPU_DEBUG 1

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer))[0]
#define FETCH_UINT32(pointer) (reinterpret_cast<unsigned int*>(&(pointer))[0])
#define FETCH_UINT32x4(pointer) (reinterpret_cast<uint4*>(&(pointer))[0])
#define FETCH_INT32(pointer) (reinterpret_cast<int*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

size_t load_from_file(char* ptr, size_t buff_size, string filepath){
    std::ifstream fin(filepath, ios::in | ios::binary);
    size_t loaded_size = fin.read(ptr, buff_size).gcount();
    return loaded_size;
}

template <
    const int BLOCK_SIZE_M, // 64
    const int BLOCK_SIZE_K, // 8
    const int BLOCK_SIZE_N, // 128
    const int THREAD_SIZE_M, // 8
    const int THREAD_SIZE_K, // 4
    const int THREAD_SIZE_N  // 8
>
__global__ void BLOCK_SPARSE_MATMUL(float* A, float* W_val, int* W_row, int* W_col, float* C, float *bias, int M, int K, int N){
    int by = blockIdx.y;
    int bx = blockIdx.x;
    int ty = threadIdx.y;
    int tx = threadIdx.x;

    __shared__ float As[BLOCK_SIZE_M * BLOCK_SIZE_K];
    __shared__ float Bs[BLOCK_SIZE_N * BLOCK_SIZE_K];

    float accum[THREAD_SIZE_N][THREAD_SIZE_M] = {0};
    float a_frag[THREAD_SIZE_M][THREAD_SIZE_K];
    float b_frag[THREAD_SIZE_N][THREAD_SIZE_K];

    int A_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    int B_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    int bszy = BLOCK_SIZE_M / THREAD_SIZE_M;
    int bszx = BLOCK_SIZE_N / THREAD_SIZE_N;

    int THREADS_PER_BLOCK = bszy * bszx;

    int A_TILE_ROW_STRIDE = THREADS_PER_BLOCK / A_THREAD_PER_ROW;
    int B_TILE_ROW_STRIDE = THREADS_PER_BLOCK / B_THREAD_PER_ROW;

    int tid = ty * bszx + tx;

    int A_BLOCK_ROW_START = tid / A_THREAD_PER_ROW;
    int B_BLOCK_ROW_START = tid / B_THREAD_PER_ROW;

    int A_BLOCK_COL_START = tid % A_THREAD_PER_ROW * 4;
    int B_BLOCK_COL_START = tid % B_THREAD_PER_ROW * 4;

    int index_start = W_row[bx], index_end = W_row[bx+1];

    const int vBLOCK_SIZE_M = BLOCK_SIZE_M / THREAD_SIZE_M;
    const int vBLOCK_SIZE_N = BLOCK_SIZE_N / THREAD_SIZE_N;
    for(int tile_block_idx = index_start; tile_block_idx < index_end; tile_block_idx += 1){
        int tile_idx = W_col[tile_block_idx] * BLOCK_SIZE_K;
        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_M; k += A_TILE_ROW_STRIDE){
            FETCH_FLOAT4(As[OFFSET(k+A_BLOCK_ROW_START, A_BLOCK_COL_START, BLOCK_SIZE_K)]) =
                FETCH_FLOAT4(A[OFFSET(by*BLOCK_SIZE_M+k+A_BLOCK_ROW_START, tile_idx+A_BLOCK_COL_START, K)]);
        }
        /*
        for(int k = 0; k < BLOCK_SIZE_K; k += A_TILE_ROW_STRIDE){
            FETCH_FLOAT4(As[OFFSET(k+A_BLOCK_ROW_START, A_BLOCK_COL_START, BLOCK_SIZE_M)]) = 
                FETCH_FLOAT4(A[OFFSET(tile_idx+k+A_BLOCK_ROW_START, by*BLOCK_SIZE_M+A_BLOCK_COL_START, M)]);
        }
        */

        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_K; k += B_TILE_ROW_STRIDE){
            FETCH_FLOAT4(Bs[OFFSET(k+B_BLOCK_ROW_START, B_BLOCK_COL_START, BLOCK_SIZE_N)]) = 
                FETCH_FLOAT4(W_val[tile_block_idx * BLOCK_SIZE_N * BLOCK_SIZE_K + (k+B_BLOCK_ROW_START) * BLOCK_SIZE_N + B_BLOCK_COL_START]);
                // FETCH_FLOAT4(B[OFFSET(tile_idx+k+B_BLOCK_ROW_START, bx*BLOCK_SIZE_N+B_BLOCK_COL_START, N)]);
        }

        __syncthreads();

        #pragma unroll
        for(int k = 0; k < BLOCK_SIZE_K; k += THREAD_SIZE_K){
            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_K; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_M; j += 1){
                    a_frag[j][i] = As[OFFSET(ty + vBLOCK_SIZE_M * j, k+i, BLOCK_SIZE_K)];
                    //a_frag[j][i] = As[OFFSET(k+i, ty + vBLOCK_SIZE_M * j, BLOCK_SIZE_M)];
                }
            }

            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_K; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_N; j += 1){
                    b_frag[j][i] = Bs[OFFSET(k+i, tx + vBLOCK_SIZE_N * j, BLOCK_SIZE_N)];
                }
            }

            #pragma unroll
            for(int i = 0; i < THREAD_SIZE_N; i++){
                #pragma unroll
                for(int j = 0; j < THREAD_SIZE_M; j++){
                    #pragma unroll
                    for(int k_in = 0; k_in < THREAD_SIZE_K; k_in++){
                        // accum[i][j] = fma(a_frag[j][k_in], b_frag[i][k_in], accum[i][j]);
                        accum[i][j] += a_frag[j][k_in] * b_frag[i][k_in];
                    }
                }
            }
        }

        __syncthreads();
    }

    float bias_local[THREAD_SIZE_N];
    for(int thread_x = 0; thread_x < THREAD_SIZE_N; thread_x++){
        bias_local[thread_x] = bias[BLOCK_SIZE_N * bx + tx + thread_x * vBLOCK_SIZE_N];
    }

    #pragma unroll
    for(int thread_x = 0; thread_x < THREAD_SIZE_N; thread_x++){
        #pragma unroll
        for(int thread_y = 0; thread_y < THREAD_SIZE_M; thread_y+=1){
            C[OFFSET(
                BLOCK_SIZE_M * by + ty + thread_y * vBLOCK_SIZE_M,
                BLOCK_SIZE_N * bx + tx + thread_x * vBLOCK_SIZE_N,
                N
            )] = (accum[thread_x][thread_y]) + bias_local[thread_x];
        }
    }
}


void HostComputation(float* A, float* W, float* D, float* bias, int M, int K, int N){
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++){
            float cSub = 0;
            for(int k = 0; k < K; k++){
                cSub += A[i * K + k] * W[k * N + j];
            }
            D[i * N + j] = cSub + bias[j];
        }
    }
}

void HostComputation_sparse(float* A, int* row, int* col, float* val, float* D, float* bias, int M, int K, int N, int BLOCK_SIZE_K, int BLOCK_SIZE_N){
    size_t mem_size_B = sizeof(float) * K * N;
    float* B = (float*)malloc(mem_size_B);
    std::memset(B, 0, sizeof(B));
    int ROW_BLOCK_NUM = K / BLOCK_SIZE_K;
    for(int i = 0; i < ROW_BLOCK_NUM; i++){
        int index_start = row[i], index_end = row[i+1];
        for(int index = index_start; index < index_end; index += 1){
            int col_index = col[index] * BLOCK_SIZE_N;
            int row_index = i * BLOCK_SIZE_K;
            float* val_ptr = val + index * BLOCK_SIZE_K * BLOCK_SIZE_N;
            for(int k = row_index; k < (i+1) * BLOCK_SIZE_K; k += 1){
                for(int n = col_index; n < col_index+BLOCK_SIZE_N; n += 1){
                    B[OFFSET(k,n,N)] = *(val_ptr + k * BLOCK_SIZE_N + n);
                }
            }
        }
    }
    for(int i = 0; i < M; i += 1){
        for(int j = 0; j < N; j += 1){
            float cSub = 0;
            for(int k = 0; k < K; k += 1){
                cSub += A[i * K + k] * B[k * N + j];
            }
            D[i * N + j] = cSub + bias[j];
        }
    }
}

int matrixMultiply(int M, int N, int K){
    int size_A = M * K;
    int size_C = M * N;

    /*
    const int BLOCK_SIZE_M = 32; // 64
    const int BLOCK_SIZE_K = 32;  //8
    const int BLOCK_SIZE_N = 32;  //128
    const int THREAD_SIZE_M = 8;  //8
    const int THREAD_SIZE_K = 4;  //4
    const int THREAD_SIZE_N = 8;  //8
    */

    const int BLOCK_SIZE_M = BLOCK_SIZE_M_VALUE; // 64
    const int BLOCK_SIZE_K = BLOCK_SIZE_K_VALUE;  //8
    const int BLOCK_SIZE_N = BLOCK_SIZE_N_VALUE;  //128
    const int THREAD_SIZE_M = THREAD_SIZE_M_VALUE;  //8
    const int THREAD_SIZE_K = THREAD_SIZE_K_VALUE;  //4
    const int THREAD_SIZE_N = THREAD_SIZE_N_VALUE;  //8

    int mem_size_A = sizeof(float) * size_A;
    int mem_size_C = sizeof(float) * size_C;
    int mem_size_bias = sizeof(float) * N;

    // memory size of row, col, val
    int mem_size_row = sizeof(int) * M;
    int mem_size_col = sizeof(int) * M * N;
    int mem_size_val = sizeof(float) * M * N;

    float* h_A = (float*)malloc(mem_size_A);
    float* h_C = (float*)malloc(mem_size_C);
    float* h_bias = (float*)malloc(mem_size_bias);
    float* h_result = (float*)malloc(mem_size_C);

    // memory allocation of row, col, val
    int* h_row = (int*)malloc(mem_size_row);
    int* h_col = (int*)malloc(mem_size_col);
    float* h_val = (float*)malloc(mem_size_val);

    // load data
    std::string row_path = ROW_PATH_VALUE;
    std::string col_path = COL_PATH_VALUE;
    std::string val_path = VAL_PATH_VALUE;

    load_from_file((char*)h_row, mem_size_row, row_path);
    load_from_file((char*)h_col, mem_size_col, col_path);
    load_from_file((char*)h_val, mem_size_val, val_path);

    float* d_A;
    float* d_C;
    float* d_bias;

    // device memory allocation
    int* d_row;
    int* d_col;
    float* d_val;

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 10;

    for(int i = 0; i < M; i++){
        for(int j = 0; j < K; j++){
            h_A[i * K + j] = rand()%5;
        }
    }

    for(int i = 0; i < N; i++){
        h_bias[i] = rand()%5;
    }

    printf("host init successfully!\n");
    printf("number of iteration: %d\n", nIter);

    checkCudaErrors(hipMalloc(&d_A, mem_size_A));
    checkCudaErrors(hipMalloc(&d_C, mem_size_C));
    checkCudaErrors(hipMalloc(&d_bias, mem_size_bias));

    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_bias, h_bias, mem_size_bias, hipMemcpyHostToDevice));

    // device csr memory copy
    checkCudaErrors(hipMalloc(&d_row, mem_size_row));
    checkCudaErrors(hipMalloc(&d_col, mem_size_col));
    checkCudaErrors(hipMalloc(&d_val, mem_size_val));

    checkCudaErrors(hipMemcpy(d_row, h_row, mem_size_row, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_col, h_col, mem_size_col, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_val, h_val, mem_size_val, hipMemcpyHostToDevice));

    printf("Device init successfully!\n");

    printf("Begin to run MatrixMulCUDA_8bit() function....\n");
    dim3 dimBlock(float(BLOCK_SIZE_N / THREAD_SIZE_N), BLOCK_SIZE_M / THREAD_SIZE_M);
    dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);

    // warm-up
    for(int run = 0; run < nIter; run++){
        BLOCK_SPARSE_MATMUL<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_M, THREAD_SIZE_K, THREAD_SIZE_N><<<dimGrid, dimBlock>>>(d_A, d_val, d_row, d_col, d_C, d_bias, M, K, N);
    }

    checkCudaErrors(hipEventRecord(start));
    for(int run = 0; run < nIter; run++) {
        BLOCK_SPARSE_MATMUL<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_M, THREAD_SIZE_K, THREAD_SIZE_N><<<dimGrid, dimBlock>>>(d_A, d_val, d_row, d_col, d_C, d_bias, M, K, N);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_result, d_C, mem_size_C, hipMemcpyDeviceToHost));

    float msecPerMatrixMul = msecTotal / nIter;

    printf("float32 block sparse kernel gemm Time= %f msec\n", msecPerMatrixMul);

#if CPU_DEBUG
    HostComputation_sparse(h_A, h_row, h_col, h_val, h_C, h_bias, M, K, N, BLOCK_SIZE_K, BLOCK_SIZE_N);
    bool correct = true;
    double eps = 1.e-4;

    for(int i = 0; i < M * N; i++){
        double abs_err = abs(h_C[i] - h_result[i]);
        double dot_length = M;
        double abs_val = abs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (abs_err > eps) {
            printf("abs_val: %lf, rel_err: %lf, abs_val: %lf, dot_length: %lf \n", abs_val, rel_err, abs_val, dot_length);
            printf("Error! Matrix[%05d]=%lf, ref=%lf error term is %lf > %E\n",
                    i, h_result[i], h_C[i], rel_err, eps);
            correct = false;
            break;
        }
    }

    if(correct) printf("Result = Pass\n");
    else printf("Result = Fail\n");
#endif
    hipFree(d_A);
    hipFree(d_C);
    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_val);

    free(h_A);
    free(h_C);
    free(h_row);
    free(h_col);
    free(h_val);

    return EXIT_SUCCESS;
}

/**
 * Program main
 */
int main(int argc, char **argv) {
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line

    int M = M_VALUE, N = N_VALUE, K = K_VALUE;

    printf("MatrixA(%d, %d), MatrixB(%d, %d)\n", M, K, K, N);

    matrixMultiply(M, N, K);
}